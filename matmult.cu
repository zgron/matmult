
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <sys/time.h>

#define M 8
#define K 8

__global__ void matmul(float *A, float *B, float *C, int N) {
  int ii = threadIdx.x + blockDim.x * blockIdx.x;
  int jj = threadIdx.y + blockDim.y * blockIdx.y;
  float sum = 0.0f;
  for (int i=ii; i<ii+M; i++) {
    for (int j=jj; j<jj+M; j++) {
      for (int k=0; k<N; k++) {
        sum += A[N*i+k] * B[N*k+j];
      }
      C[N*i+j] = sum;
    }
  }
}

int main(int argc, char **argv) {
  int N = atoi(argv[1]);
  float * h_A = new float [N*N];
  float * h_B = new float [N*N];
  float * h_C = new float [N*N];
  float *d_A, *d_B, *d_C;
  int size = N * N * sizeof(float);
  hipMalloc((void **) &d_A, size);
  hipMalloc((void **) &d_B, size);
  hipMalloc((void **) &d_C, size);

  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      h_A[N*i+j] = drand48();
      h_B[N*i+j] = drand48();
      h_C[N*i+j] = 0;
    }
  }
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
  dim3 grid(N/M/K, N/M/K);
  dim3 block(K,K);
  struct timeval tic, toc;
  gettimeofday(&tic, NULL);
  matmul<<<grid,block>>>(d_A, d_B, d_C, N);
  hipDeviceSynchronize();
  gettimeofday(&toc, NULL);
  double time = toc.tv_sec-tic.tv_sec+(toc.tv_usec-tic.tv_usec)*1e-6;
  printf("N=%d: %lf s (%lf GFlops)\n",N,time,2.*N*N*N/time/1e9);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  gettimeofday(&tic, NULL);
#pragma omp parallel for
  for (int i=0; i<N; i++) {
    for (int k=0; k<N; k++) {
      for (int j=0; j<N; j++) {
        h_C[N*i+j] -= h_A[N*i+k] * h_B[N*k+j];
      }
    }
  }
  gettimeofday(&toc, NULL);
  time = toc.tv_sec-tic.tv_sec+(toc.tv_usec-tic.tv_usec)*1e-6;
  printf("N=%d: %lf s (%lf GFlops)\n",N,time,2.*N*N*N/time/1e9);
  float err = 0;
  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      err += fabs(h_C[N*i+j]);
    }
  }
  printf("error: %f\n",err/N/N);
  delete[] h_A;
  delete[] h_B;
  delete[] h_C;
}
