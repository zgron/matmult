
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <sys/time.h>

#define M 1024

__global__ void matmul(float *A, float *B, float *C, int N) {
  int i = blockIdx.y;
  int j = threadIdx.x + blockDim.x * blockIdx.x;
  float sum = 0.0f;
  __shared__ float s_A[M];
  for (int ks=0; ks<N; ks+=M) {
    __syncthreads();
    s_A[threadIdx.x] = A[N*i+ks+threadIdx.x];
    __syncthreads();
    for (int k=ks; k<ks+M; k++) {
      sum += s_A[k-ks] * B[N*k+j];
    }
  }
  C[N*i+j] = sum;
}

int main(int argc, char **argv) {
  int N = atoi(argv[1]);
  float * h_A = new float [N*N];
  float * h_B = new float [N*N];
  float * h_C = new float [N*N];
  float *d_A, *d_B, *d_C;
  int size = N * N * sizeof(float);
  hipMalloc((void **) &d_A, size);
  hipMalloc((void **) &d_B, size);
  hipMalloc((void **) &d_C, size);

  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      h_A[N*i+j] = drand48();
      h_B[N*i+j] = drand48();
      h_C[N*i+j] = 0;
    }
  }
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
  dim3 grid(N/M, N);
  struct timeval tic, toc;
  gettimeofday(&tic, NULL);
  matmul<<<grid,M>>>(d_A, d_B, d_C, N);
  hipDeviceSynchronize();
  gettimeofday(&toc, NULL);
  double time = toc.tv_sec-tic.tv_sec+(toc.tv_usec-tic.tv_usec)*1e-6;
  printf("N=%d: %lf s (%lf GFlops)\n",N,time,2.*N*N*N/time/1e9);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  gettimeofday(&tic, NULL);
#pragma omp parallel for
  for (int i=0; i<N; i++) {
    for (int k=0; k<N; k++) {
      for (int j=0; j<N; j++) {
        h_C[N*i+j] -= h_A[N*i+k] * h_B[N*k+j];
      }
    }
  }
  gettimeofday(&toc, NULL);
  time = toc.tv_sec-tic.tv_sec+(toc.tv_usec-tic.tv_usec)*1e-6;
  printf("N=%d: %lf s (%lf GFlops)\n",N,time,2.*N*N*N/time/1e9);
  float err = 0;
  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      err += fabs(h_C[N*i+j]);
    }
  }
  printf("error: %f\n",err/N/N);
  delete[] h_A;
  delete[] h_B;
  delete[] h_C;
}
